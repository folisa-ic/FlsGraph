#include "hip/hip_runtime.h"
#include "head.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
#include <time.h>

// ��ʼ��GPU
bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);		// ȡ��֧�� CUDA ���豸����Ŀ�����ϵͳ��û��֧��CUDA��װ�ã���ᴫ��1
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {	// device0��һ�������豸����֧��CUDA1.0���ϵĹ���
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {		// hipGetDeviceProperties��ȡ��װ�õĸ������ݣ�������Ϊ�˻�ȡ֧�ְ汾��prop.major��
			if (prop.major >= 1) {
				break;				// �����֧��CUDA1.0���ϰ汾���豸������
			}
		}
	}
	if (i == count) {				// ��ʾû���ҵ�֧��CUDA1.0���ϰ汾���豸
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);				// ���ҵ����豸��Ϊ��ǰʹ�õ��豸
	return true;
}

// GPU kernal����scatter
__global__ void scatter_and_gather(
	float* dest_vertex_weight_device,
	float* msg_value_device,
	float* dest_vertex_weight_gathered_device,
	struct edge_in_GPU* edge_in_GPU_struct)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	msg_value_device[bid * NUM_EDGE_LIST + tid] = edge_in_GPU_struct[bid * NUM_EDGE_LIST + tid].edge_weight + edge_in_GPU_struct[bid * NUM_EDGE_LIST + tid].src_vertex_weight;
	if (msg_value_device[bid * NUM_EDGE_LIST + tid] < dest_vertex_weight_device[bid * NUM_EDGE_LIST + tid])
	{
		dest_vertex_weight_gathered_device[bid * NUM_EDGE_LIST + tid] = msg_value_device[bid * NUM_EDGE_LIST + tid];
		// __syncthreads();
	}
	else
	{
		dest_vertex_weight_gathered_device[bid * NUM_EDGE_LIST + tid] = 10000;
		// __syncthreads();
	}
}

extern "C" void GPU_KERNEL(
	float* dest_vertex_weight_host,
	float* dest_vertex_weight_gathered_host,
	float* time_for_single_loop,
	edge_in_CPU_P * edge_in_CPU_struct,
	Edge_Set_P * Edge_Set
)
{
	float* msg_value_device;
	float* dest_vertex_weight_gathered_device;
	float* dest_vertex_weight_device;

	hipMalloc(&msg_value_device, K * NUM_EDGE_LIST * sizeof(float));
	hipMalloc(&dest_vertex_weight_device, K * NUM_EDGE_LIST * sizeof(float));
	hipMalloc(&dest_vertex_weight_gathered_device, K * NUM_EDGE_LIST * sizeof(float));

	struct edge_in_GPU* edge_in_GPU_struct;
	// ���Դ��������ṹ��ָ�������ռ�
	hipMalloc(&edge_in_GPU_struct, K * NUM_EDGE_LIST * sizeof(edge_in_GPU));

	for (int i = 0; i < K; i++)
	{
		int num_edge = Edge_Set[i]->num_edge;

		// ��CPU�ṹ���ڵ�����ת�Ƶ���������host������
		for (int j = 0; j < num_edge; j++)
		{
			int num = i * NUM_EDGE_LIST + j;
			hipMemcpy(&edge_in_GPU_struct[num], edge_in_CPU_struct[num], sizeof(struct edge_in_CPU), hipMemcpyHostToDevice);
		}
	}

	// �������ݵ�GPU�ڴ���
	hipMemcpy(dest_vertex_weight_device, dest_vertex_weight_host, K * NUM_EDGE_LIST * sizeof(float), hipMemcpyHostToDevice);

	InitCUDA();

	hipEvent_t start, stop;
	float time = 0.f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	scatter_and_gather << <K, NUM_EDGE_LIST >> > (
		dest_vertex_weight_device,
		msg_value_device,
		dest_vertex_weight_gathered_device,
		edge_in_GPU_struct
		);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	*time_for_single_loop = time;
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// ������������CPU�����ڽ��յ��ڴ���
	hipMemcpy(dest_vertex_weight_gathered_host, dest_vertex_weight_gathered_device, K * NUM_EDGE_LIST * sizeof(float), hipMemcpyDeviceToHost);

	// �ͷŵ�GPU�ڴ�
	hipFree(msg_value_device);
	hipFree(dest_vertex_weight_device);
	hipFree(dest_vertex_weight_gathered_device);
}