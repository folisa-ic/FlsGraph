#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "head.h"

// ��ʼ��GPU
bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);		// ȡ��֧�� CUDA ���豸����Ŀ�����ϵͳ��û��֧��CUDA��װ�ã���ᴫ��1
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {	// device0��һ�������豸����֧��CUDA1.0���ϵĹ���
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {		// hipGetDeviceProperties��ȡ��װ�õĸ������ݣ�������Ϊ�˻�ȡ֧�ְ汾��prop.major��
			if (prop.major >= 1) {
				break;				// �����֧��CUDA1.0���ϰ汾���豸������
			}
		}
	}
	if (i == count) {				// ��ʾû���ҵ�֧��CUDA1.0���ϰ汾���豸
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);				// ���ҵ����豸��Ϊ��ǰʹ�õ��豸
	return true;
}

// GPU kernal����scatter
__global__ void scatter_and_gather(
	float* edge_weight_device,
	float* src_vertex_weight_device,
	float* dest_vertex_weight_device,
	float* msg_value_device,
	float* dest_vertex_weight_gathered_device)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	msg_value_device[bid * NUM_EDGE_LIST + tid] = edge_weight_device[bid * NUM_EDGE_LIST + tid] + src_vertex_weight_device[bid * NUM_EDGE_LIST + tid];
	if (msg_value_device[bid * NUM_EDGE_LIST + tid] < dest_vertex_weight_device[bid * NUM_EDGE_LIST + tid])
	{
		dest_vertex_weight_gathered_device[bid * NUM_EDGE_LIST + tid] = msg_value_device[bid * NUM_EDGE_LIST + tid];
		// __syncthreads();
	}
	else
	{
		dest_vertex_weight_gathered_device[bid * NUM_EDGE_LIST + tid] = 10000;
		// __syncthreads();
	}
}

extern "C" void GPU_KERNEL(
	float* edge_weight_host,
	float* src_vertex_weight_host,
	float* dest_vertex_weight_host,
	float* dest_vertex_weight_gathered_host,
	float* time_for_single_loop
)
{
	float* msg_value_device;
	float* dest_vertex_weight_gathered_device;
	float* edge_weight_device;
	float* src_vertex_weight_device;
	float* dest_vertex_weight_device;

	hipMalloc(&msg_value_device, K * NUM_EDGE_LIST * sizeof(float));
	hipMalloc(&edge_weight_device, K * NUM_EDGE_LIST * sizeof(float));
	hipMalloc(&src_vertex_weight_device, K * NUM_EDGE_LIST * sizeof(float));
	hipMalloc(&dest_vertex_weight_device, K * NUM_EDGE_LIST * sizeof(float));
	hipMalloc(&dest_vertex_weight_gathered_device, K * NUM_EDGE_LIST * sizeof(float));

	// �������ݵ�GPU�ڴ���
	hipMemcpy(edge_weight_device, edge_weight_host, K * NUM_EDGE_LIST * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(src_vertex_weight_device, src_vertex_weight_host, K * NUM_EDGE_LIST * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dest_vertex_weight_device, dest_vertex_weight_host, K * NUM_EDGE_LIST * sizeof(float), hipMemcpyHostToDevice);

	InitCUDA();

	hipEvent_t start, stop;
	float time = 0.f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	scatter_and_gather << <K, NUM_EDGE_LIST >> > (
		edge_weight_device,
		src_vertex_weight_device,
		dest_vertex_weight_device,
		msg_value_device,
		dest_vertex_weight_gathered_device
		);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	*time_for_single_loop = time;
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// ������������CPU�����ڽ��յ��ڴ���
	hipMemcpy(dest_vertex_weight_gathered_host, dest_vertex_weight_gathered_device, K * NUM_EDGE_LIST * sizeof(float), hipMemcpyDeviceToHost);

	// �ͷŵ�GPU�ڴ�
	hipFree(msg_value_device);
	hipFree(edge_weight_device);
	hipFree(src_vertex_weight_device);
	hipFree(dest_vertex_weight_device);
	hipFree(dest_vertex_weight_gathered_device);
}